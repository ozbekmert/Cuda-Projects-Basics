#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<stdio.h>


__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU! \n");
}

int main()
{
	helloFromGPU <<<1, 10 >> >();
	hipDeviceReset();
	return 0;
}
